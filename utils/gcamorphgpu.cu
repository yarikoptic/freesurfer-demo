
#include <hip/hip_runtime.h>
/**
 * @file  gcamorphgpu.cu
 * @brief Holds GCA morph data on the GPU
 *
 * 
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2010/10/22 22:40:14 $
 *    $Revision: 1.36.2.1 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 *
 */

#ifdef GCAMORPH_ON_GPU

#include "macros.h"

#include "chronometer.hpp"

#include "volumegpucompare.hpp"


#include "gcamorphgpu.hpp"


//! Texture reference for rx
texture<float,3,cudaReadModeElementType> dt_rx;
//! Texture reference for ry
texture<float,3,cudaReadModeElementType> dt_ry;
//! Texture reference for rz
texture<float,3,cudaReadModeElementType> dt_rz;

//! Texture reference for dx
texture<float,3,cudaReadModeElementType> dt_dx;
//! Texture reference for dy
texture<float,3,cudaReadModeElementType> dt_dy;
//! Texture reference for dz
texture<float,3,cudaReadModeElementType> dt_dz;


// ==============================================================

namespace GPU {
  namespace Classes {

    // --------------------------------------------

    void GCAmorphGPU::CheckIntegrity( void ) const {
      /*!
	Checks that all the allocated member arrays have
	the same dimensions.
	Aborts the program if the check fails
      */

      const dim3 myDims = this->d_rx.GetDims();

      bool good = ( myDims == this->d_ry.GetDims() );
      good = ( good && ( myDims == this->d_rz.GetDims() ) );

      good = ( good && ( myDims == this->d_origx.GetDims() ) );
      good = ( good && ( myDims == this->d_origy.GetDims() ) );
      good = ( good && ( myDims == this->d_origz.GetDims() ) );

      good = ( good && ( myDims == this->d_dx.GetDims() ) );
      good = ( good && ( myDims == this->d_dy.GetDims() ) );
      good = ( good && ( myDims == this->d_dz.GetDims() ) );
      
      good = ( good && ( myDims == this->d_odx.GetDims() ) );
      good = ( good && ( myDims == this->d_ody.GetDims() ) );
      good = ( good && ( myDims == this->d_odz.GetDims() ) );

      good = ( good && ( myDims == this->d_invalid.GetDims() ) );

      good = ( good && ( myDims == this->d_origArea.GetDims() ) );
      good = ( good && ( myDims == this->d_origArea1.GetDims() ) );
      good = ( good && ( myDims == this->d_origArea2.GetDims() ) );

      good = ( good && ( myDims == this->d_area.GetDims() ) );
      good = ( good && ( myDims == this->d_area1.GetDims() ) );
      good = ( good && ( myDims == this->d_area2.GetDims() ) );

      good = ( good && ( myDims == this->d_label.GetDims() ) );
      good = ( good && ( myDims == this->d_status.GetDims() ) );

      good = ( good && ( myDims == this->d_mean.GetDims() ) );
      good = ( good && ( myDims == this->d_variance.GetDims() ) );

      good = ( good && ( myDims == this->d_labelDist.GetDims() ) );

      if( !good ) {
	std::cerr << __FUNCTION__
		  << ": Dimension mismatch"
		  << std::endl;
	exit( EXIT_FAILURE );
      }
    }

    // --------------------------------------------

    void GCAmorphGPU::AllocateAll( const dim3& dims ) {
      /*!
	Allocates GPU memory to hold a volume
	of the given size.
	If possible, it keeps the current allocation.
      */

      // Start by seeing if the current allocation is consistent
      this->CheckIntegrity();

      // See if we can re-use existing allocation
      if( dims == this->d_rx.GetDims() ) {
	return;
      }

      // Release existing memory
      this->ReleaseAll();

      // Allocate anew
      this->d_rx.Allocate( dims );
      this->d_ry.Allocate( dims );
      this->d_rz.Allocate( dims );

      this->d_origx.Allocate( dims );
      this->d_origy.Allocate( dims );
      this->d_origz.Allocate( dims );
 
      this->d_dx.Allocate( dims );
      this->d_dy.Allocate( dims );
      this->d_dz.Allocate( dims );
 
      this->d_odx.Allocate( dims );
      this->d_ody.Allocate( dims );
      this->d_odz.Allocate( dims );

      this->d_area.Allocate( dims );
      this->d_area1.Allocate( dims );
      this->d_area2.Allocate( dims );

      this->d_origArea.Allocate( dims );
      this->d_origArea1.Allocate( dims );
      this->d_origArea2.Allocate( dims );

      this->d_invalid.Allocate( dims );
      this->d_label.Allocate( dims );
      this->d_status.Allocate( dims );
      this->d_labelDist.Allocate( dims );

      this->d_mean.Allocate( dims );
      this->d_variance.Allocate( dims );
    }


    void GCAmorphGPU::ReleaseAll( void ) {
      /*!
	Releases each of the members.
	Recall that the VolumeGPU::Release method
	will also release any CUDA arrays.
      */
      this->d_rx.Release();
      this->d_ry.Release();
      this->d_rz.Release();

      this->d_dx.Release();
      this->d_dy.Release();
      this->d_dz.Release();

      this->d_odx.Release();
      this->d_ody.Release();
      this->d_odz.Release();

      this->d_origx.Release();
      this->d_origy.Release();
      this->d_origz.Release();

      this->d_origArea.Release();
      this->d_origArea1.Release();
      this->d_origArea2.Release();

      this->d_area.Release();
      this->d_area1.Release();
      this->d_area2.Release();

      this->d_invalid.Release();
      this->d_label.Release();
      this->d_status.Release();
      this->d_labelDist.Release();

      this->d_mean.Release();
      this->d_variance.Release();

    }



    void GCAmorphGPU::ClearAll( void ) {
      this->d_rx.Zero();
      this->d_ry.Zero();
      this->d_rz.Zero();

      this->d_dx.Zero();
      this->d_dy.Zero();
      this->d_dz.Zero();

      this->d_odx.Zero();
      this->d_ody.Zero();
      this->d_odz.Zero();

      this->d_origx.Zero();
      this->d_origy.Zero();
      this->d_origz.Zero();

      this->d_origArea.Zero();
      this->d_origArea1.Zero();
      this->d_origArea2.Zero();

      this->d_area.Zero();
      this->d_area1.Zero();
      this->d_area2.Zero();

      this->d_invalid.Zero();
      this->d_label.Zero();
      this->d_status.Zero();
      this->d_labelDist.Zero();

      this->d_mean.Zero();
      this->d_variance.Zero();
    }


    // --------------------------------------------

    void GCAmorphGPU::SendAll( const GCAM* src ) {
      /*!
	Sends all supported data in the given GCAM
	to the GPU.
	This involves a lot of packing data, and hence
	is going to be painfully slow
      */

      GCAmorphGPU::tSendTot.Start();

#if 0
      std::cerr << __FUNCTION__
		<< ": Catching gcamorph usage"
		<< std::endl;
      exit( EXIT_FAILURE );
#endif

      // Check for number of inputs
      if( src->ninputs != 1 ) {
	std::cerr << __FUNCTION__
		  << ": Must have only one input in the GC1D!"
		  << std::endl;
	exit( EXIT_FAILURE );
      }

      // Copy scalars
      this->exp_k = src->exp_k;
      this->neg = src->neg;
      this->gca = src->gca;

      // Extract the dimensions
      const dim3 dims = make_uint3( src->width,
				    src->height,
				    src->depth );

      // Allocate device memory
      this->AllocateAll( dims );

      // Allocate some page-locked host buffers
      GCAmorphGPU::AllocateHost( *this );


      GCAmorphGPU::tSendPack.Start();
      for( unsigned int i=0; i<dims.x; i++ ) {
	for( unsigned int j=0; j<dims.y; j++ ) {
	  for( unsigned int k=0; k<dims.z; k++ ) {

	    // Get the 1d index (same for all arrays)
	    const unsigned int i1d = this->d_rx.Index1D( i, j, k );
	    // Get the current node
	    const GCA_MORPH_NODE& gcamn = src->nodes[i][j][k];
	    
	    // Pack the data
	    GCAmorphGPU::h_rx[i1d] = gcamn.x;
	    GCAmorphGPU::h_ry[i1d] = gcamn.y;
	    GCAmorphGPU::h_rz[i1d] = gcamn.z;

	    GCAmorphGPU::h_origx[i1d] = gcamn.origx;
	    GCAmorphGPU::h_origy[i1d] = gcamn.origy;
	    GCAmorphGPU::h_origz[i1d] = gcamn.origz;

	    GCAmorphGPU::h_dx[i1d] = gcamn.dx;
	    GCAmorphGPU::h_dy[i1d] = gcamn.dy;
	    GCAmorphGPU::h_dz[i1d] = gcamn.dz;

	    GCAmorphGPU::h_odx[i1d] = gcamn.odx;
	    GCAmorphGPU::h_ody[i1d] = gcamn.ody;
	    GCAmorphGPU::h_odz[i1d] = gcamn.odz;

	    GCAmorphGPU::h_origArea[i1d] = gcamn.orig_area;
	    GCAmorphGPU::h_origArea1[i1d] = gcamn.orig_area1;
	    GCAmorphGPU::h_origArea2[i1d] = gcamn.orig_area2;

	    GCAmorphGPU::h_area[i1d] = gcamn.area;
	    GCAmorphGPU::h_area1[i1d] = gcamn.area1;
	    GCAmorphGPU::h_area2[i1d] = gcamn.area2;

	    GCAmorphGPU::h_invalid[i1d] = gcamn.invalid;
	    GCAmorphGPU::h_status[i1d] = gcamn.status;
	    GCAmorphGPU::h_label[i1d] = gcamn.label;
	    GCAmorphGPU::h_labelDist[i1d] = gcamn.label_dist;

	    // Deal with the GC1D
	    if( gcamn.gc != NULL ) {
	      /*
		Store the mean and variance.
		Check at top of the routine has ensured
		that there's only one input.
		This means that the covariance is really
		a variance
	      */
	      GCAmorphGPU::h_mean[i1d] = gcamn.gc->means[0];
	      GCAmorphGPU::h_variance[i1d] = gcamn.gc->covars[0];
	    } else {
	      /*
		Store negative numbers to indicate that
		there is no GC1D here.
		Since a variance must be >=0, this is
		a reliable test
	      */
	      GCAmorphGPU::h_mean[i1d] = -1;
	      GCAmorphGPU::h_variance[i1d] = -1;
	    }


	  }
	}
      }
      GCAmorphGPU::tSendPack.Stop();


      GCAmorphGPU::tSendTransfer.Start();
      // Send the data
      this->d_rx.SendBuffer( GCAmorphGPU::h_rx );
      this->d_ry.SendBuffer( GCAmorphGPU::h_ry );
      this->d_rz.SendBuffer( GCAmorphGPU::h_rz );

      this->d_origx.SendBuffer( GCAmorphGPU::h_origx );
      this->d_origy.SendBuffer( GCAmorphGPU::h_origy );
      this->d_origz.SendBuffer( GCAmorphGPU::h_origz );

      this->d_dx.SendBuffer( GCAmorphGPU::h_dx );
      this->d_dy.SendBuffer( GCAmorphGPU::h_dy );
      this->d_dz.SendBuffer( GCAmorphGPU::h_dz );

      this->d_odx.SendBuffer( GCAmorphGPU::h_odx );
      this->d_ody.SendBuffer( GCAmorphGPU::h_ody );
      this->d_odz.SendBuffer( GCAmorphGPU::h_odz );
      
      this->d_origArea.SendBuffer( GCAmorphGPU::h_origArea );
      this->d_origArea1.SendBuffer( GCAmorphGPU::h_origArea1 );
      this->d_origArea2.SendBuffer( GCAmorphGPU::h_origArea2 );

      this->d_area.SendBuffer( GCAmorphGPU::h_area );
      this->d_area1.SendBuffer( GCAmorphGPU::h_area1 );
      this->d_area2.SendBuffer( GCAmorphGPU::h_area2 );

      this->d_invalid.SendBuffer( GCAmorphGPU::h_invalid );
      this->d_status.SendBuffer( GCAmorphGPU::h_status );
      this->d_label.SendBuffer( GCAmorphGPU::h_label );
      this->d_labelDist.SendBuffer( GCAmorphGPU::h_labelDist );

      this->d_mean.SendBuffer( GCAmorphGPU::h_mean );
      this->d_variance.SendBuffer( GCAmorphGPU::h_variance );

      // Wait for the copies to complete
      CUDA_SAFE_CALL( cudaThreadSynchronize() );
      GCAmorphGPU::tSendTransfer.Stop();


      GCAmorphGPU::tSendTot.Stop();

    }

    // --------------------------------------------

    void GCAmorphGPU::RecvAll( GCAM* dst ) const {
      /*!
	Retrieves all supported data in the given GCAM
	from the GPU.
	This involves a lot of packing data, and hence
	is going to be painfully slow
      */

      GCAmorphGPU::tRecvTot.Start();

      // Check for number of inputs
      if( dst->ninputs != 1 ) {
	std::cerr << __FUNCTION__
		  << ": Must have only one input in the GC1D!"
		  << std::endl;
	exit( EXIT_FAILURE );
      }


      // Copy scalars
      dst->exp_k = this->exp_k;
      dst->neg = this->neg;
      std::cerr << __FUNCTION__
		<< ": Did not reset gca in dst"
		<< std::endl;

      // Extract the dimensions
      const dim3 dims = this->d_rx.GetDims();

      // Allocate page-locked host memory
      GCAmorphGPU::AllocateHost( *this );

      GCAmorphGPU::tRecvTransfer.Start();
      // Fetch the data
      this->d_rx.RecvBuffer( GCAmorphGPU::h_rx );
      this->d_ry.RecvBuffer( GCAmorphGPU::h_ry );
      this->d_rz.RecvBuffer( GCAmorphGPU::h_rz );

      this->d_origx.RecvBuffer( GCAmorphGPU::h_origx );
      this->d_origy.RecvBuffer( GCAmorphGPU::h_origy );
      this->d_origz.RecvBuffer( GCAmorphGPU::h_origz );

      this->d_dx.RecvBuffer( GCAmorphGPU::h_dx );
      this->d_dy.RecvBuffer( GCAmorphGPU::h_dy );
      this->d_dz.RecvBuffer( GCAmorphGPU::h_dz );

      this->d_odx.RecvBuffer( GCAmorphGPU::h_odx );
      this->d_ody.RecvBuffer( GCAmorphGPU::h_ody );
      this->d_odz.RecvBuffer( GCAmorphGPU::h_odz );

      this->d_origArea.RecvBuffer( GCAmorphGPU::h_origArea );
      this->d_origArea1.RecvBuffer( GCAmorphGPU::h_origArea1 );
      this->d_origArea2.RecvBuffer( GCAmorphGPU::h_origArea2 );

      this->d_area.RecvBuffer( GCAmorphGPU::h_area );
      this->d_area1.RecvBuffer( GCAmorphGPU::h_area1 );
      this->d_area2.RecvBuffer( GCAmorphGPU::h_area2 );

      this->d_invalid.RecvBuffer( GCAmorphGPU::h_invalid );
      this->d_status.RecvBuffer( GCAmorphGPU::h_status );
      this->d_label.RecvBuffer( GCAmorphGPU::h_label );
      this->d_labelDist.RecvBuffer( GCAmorphGPU::h_labelDist );

      this->d_mean.RecvBuffer( GCAmorphGPU::h_mean );
      this->d_variance.RecvBuffer( GCAmorphGPU:: h_variance );
      CUDA_SAFE_CALL( cudaThreadSynchronize() );
      GCAmorphGPU::tRecvTransfer.Stop();

      GCAmorphGPU::tRecvPack.Start();
      for( unsigned int i=0; i<dims.x; i++ ) {
	for( unsigned int j=0; j<dims.y; j++ ) {
	  for( unsigned int k=0; k<dims.z; k++ ) {

	    // Get the 1d index (same for all arrays)
	    const unsigned int i1d = this->d_rx.Index1D( i, j, k );
	    // Get the current node
	    GCA_MORPH_NODE* gcamn = &(dst->nodes[i][j][k]);

	    gcamn->x = GCAmorphGPU::h_rx[i1d];
	    gcamn->y = GCAmorphGPU::h_ry[i1d];
	    gcamn->z = GCAmorphGPU::h_rz[i1d];

	    gcamn->origx = GCAmorphGPU::h_origx[i1d];
	    gcamn->origy = GCAmorphGPU::h_origy[i1d];
	    gcamn->origz = GCAmorphGPU::h_origz[i1d];
	    
	    gcamn->dx = GCAmorphGPU::h_dx[i1d];
	    gcamn->dy = GCAmorphGPU::h_dy[i1d];
	    gcamn->dz = GCAmorphGPU::h_dz[i1d];
	    
	    gcamn->odx = GCAmorphGPU::h_odx[i1d];
	    gcamn->ody = GCAmorphGPU::h_ody[i1d];
	    gcamn->odz = GCAmorphGPU::h_odz[i1d];

	    gcamn->orig_area = GCAmorphGPU::h_origArea[i1d];
	    gcamn->orig_area1 = GCAmorphGPU::h_origArea1[i1d];
	    gcamn->orig_area2 = GCAmorphGPU::h_origArea2[i1d];

	    gcamn->area = GCAmorphGPU::h_area[i1d];
	    gcamn->area1 = GCAmorphGPU::h_area1[i1d];
	    gcamn->area2 = GCAmorphGPU::h_area2[i1d];

	    gcamn->invalid = GCAmorphGPU::h_invalid[i1d];
	    gcamn->label = GCAmorphGPU::h_label[i1d];
	    gcamn->label_dist = GCAmorphGPU::h_labelDist[i1d];
	    gcamn->status = GCAmorphGPU::h_status[i1d];

	    // We now have a quandary... how to test for validity
	    if( gcamn->gc != NULL ) {
	      // We know there's only one input from test at the top
	      gcamn->gc->means[0] = GCAmorphGPU::h_mean[i1d];
	      gcamn->gc->covars[0] = GCAmorphGPU::h_variance[i1d];
	    } else {
	      if( GCAmorphGPU::h_variance[i1d] >= 0 ) {
		std::cerr << __FUNCTION__
			  << ": Host has no GC1D but GPU has valid variance"
			  << std::endl;
		exit( EXIT_FAILURE );
	      }
	    }

	  }
	}
      }
      GCAmorphGPU::tRecvPack.Stop();

      GCAmorphGPU::tRecvTot.Stop();

    }




    // --------------------------------------------

    const unsigned int kCMPKernelSize = 16;
    const unsigned int iCMPGlobalsInvalid = 0;
    const unsigned int iCMPGlobalsNeg = 1;

    //! Device function to look up displacement vectors
    __device__ float3 FetchVector( const unsigned int ix,
				   const unsigned int iy,
				   const unsigned int iz ) {

      float3 r;
      r.x = tex3D( dt_rx, ix+0.5f, iy+0.5f, iz+0.5f );
      r.y = tex3D( dt_ry, ix+0.5f, iy+0.5f, iz+0.5f );
      r.z = tex3D( dt_rz, ix+0.5f, iy+0.5f, iz+0.5f );

      return( r );
    }

    //! Kernel to perform work of gcamComputeMetricProperties
    __global__
    void CompMetPropKernel( const VolumeArgGPU<float> origArea,
			    VolumeArgGPU<char> invalid,
			    VolumeArgGPU<float> area,
			    VolumeArgGPU<float> area1,
			    VolumeArgGPU<float> area2,
			    int *globals ) {
      /*!
	This kernel performs the work of gcamComputeMetricProperties.
	For now, it's unoptimised, and may cause a lot of un-necessary
	memory transations
      */
      // Compute co-ordinates
      const unsigned int ix = threadIdx.x + ( blockIdx.x * blockDim.x );
      const unsigned int iy = threadIdx.y + ( blockIdx.y * blockDim.y );

      // Check if in volume
      if( !origArea.InVolume( ix, iy, 0 ) ) {
	return;
      }

      // Loop over each z slice
      for( unsigned int iz=0; iz< origArea.dims.z; iz++ ) {

	int neg = 0;
	int num = 0;

	// Check for invalid node
	if( invalid( ix, iy, iz ) == GCAM_POSITION_INVALID ) {
	  atomicAdd( &(globals[iCMPGlobalsInvalid]), 1 );
	  continue;
	}

	// Fetch the location of the current voxel
	const float3 r = FetchVector( ix, iy, iz );
	
	// Zero the 'area'
	area(ix,iy,iz) = 0;

	// Compute Jacobean determinants on the 'right'
	if( (ix<origArea.dims.x-1) &&
	    (iy<origArea.dims.y-1) &&
	    (iz<origArea.dims.z-1) ) {


	  // Check for validity
	  if( (invalid(ix+1,iy,iz) != GCAM_POSITION_INVALID) &&
	      (invalid(ix,iy+1,iz) != GCAM_POSITION_INVALID) &&
	      (invalid(ix,iy,iz+1) != GCAM_POSITION_INVALID) ) {
	    
	    num++;
	    

	    float3 vi = FetchVector(ix+1,iy  ,iz  ) - r;
	    float3 vj = FetchVector(ix  ,iy+1,iz  ) - r;
	    float3 vk = FetchVector(ix  ,iy  ,iz+1) - r;

	    float tmpArea = stp( vj, vk, vi );
	    if( tmpArea <= 0 ) {
	      neg = 1;
	    }

	    area1(ix,iy,iz) = tmpArea;
	    area(ix,iy,iz) += tmpArea;

	  }
	} else {
	  // Going to 'right' would fall out of the volume
	  area1(ix,iy,iz) = 0;
	}


	// Compute Jacobean determinants on the 'left'
	if( (ix>0) && (iy>0) && (iz>0) ) {
	  
	  // Check for validity
	  if( (invalid(ix-1,iy,iz) != GCAM_POSITION_INVALID) &&
	      (invalid(ix,iy-1,iz) != GCAM_POSITION_INVALID) &&
	      (invalid(ix,iy,iz-1) != GCAM_POSITION_INVALID) ) {
	    num++;

	    // I think this ordering preserves handedness
	    // It's different to that in gcamorph.c
	    float3 vi = r - FetchVector(ix-1,iy  ,iz  );
	    float3 vj = r - FetchVector(ix  ,iy-1,iz  );
	    float3 vk = r - FetchVector(ix  ,iy  ,iz-1);

	    float tmpArea = stp( vj, vk, vi );

	    if( tmpArea <= 0 ) {
	      neg = 1;
	    }

	    area2(ix,iy,iz) = tmpArea;
	    area(ix,iy,iz) += tmpArea;
	  }
	} else {
	  area2(ix,iy,iz) = 0;
	}

	// Check if at least one determinant was computed
	if( num > 0 ) {
	  // area is mean of 'left' and 'right' areas
	  area(ix,iy,iz) /= num;
	} else {
	  invalid(ix,iy,iz) = GCAM_AREA_INVALID;
	  area(ix,iy,iz) = 0;
	}

	// Keep track of sign changes
	if( (invalid(ix,iy,iz)==GCAM_VALID) &&
	    neg &&
	    origArea(ix,iy,iz) > 0 ) {
	  atomicAdd( &(globals[iCMPGlobalsNeg]), 1 );
	}

	// Increment invalid counter
	if( invalid(ix,iy,iz) != GCAM_VALID ) {
	  // We need to test again
	  atomicAdd( &(globals[iCMPGlobalsInvalid]), 1 );
	}
      }
    }
    
    void GCAmorphGPU::ComputeMetricProperties( int& invalid ) {
      /*!
	Routine to duplicate gcamComputeMetricProperties
	from the file gcamorph.c.
	It essentially computes a lot of jacobean determinants
	and sums them up.
	The argument \a invalid is used to return the number of
	invalid locations found, a task performed by the
	global variable \c Ginvalid in gcamorph.c.
      */


      GCAmorphGPU::tCMPtot.Start();

      // Sanity check
      this->CheckIntegrity();

      // Allocate temporary on the device to hold invalid and neg
      int *d_globals;
      CUDA_SAFE_CALL( cudaMalloc( (void**)&d_globals, 2*sizeof(int) ) );
      CUDA_SAFE_CALL( cudaMemset( d_globals, 0, 2*sizeof(int) ) );

      // Get the d_rx, d_ry and d_rz fields bound to textures
      this->d_rx.AllocateArray();
      this->d_ry.AllocateArray();
      this->d_rz.AllocateArray();
      this->d_rx.SendArray();
      this->d_ry.SendArray();
      this->d_rz.SendArray();

      dt_rx.normalized = false;
      dt_rx.addressMode[0] = cudaAddressModeClamp;
      dt_rx.addressMode[1] = cudaAddressModeClamp;
      dt_rx.addressMode[2] = cudaAddressModeClamp;
      dt_rx.filterMode = cudaFilterModePoint;

      dt_ry.normalized = false;
      dt_ry.addressMode[0] = cudaAddressModeClamp;
      dt_ry.addressMode[1] = cudaAddressModeClamp;
      dt_ry.addressMode[2] = cudaAddressModeClamp;
      dt_ry.filterMode = cudaFilterModePoint;

      dt_rz.normalized = false;
      dt_rz.addressMode[0] = cudaAddressModeClamp;
      dt_rz.addressMode[1] = cudaAddressModeClamp;
      dt_rz.addressMode[2] = cudaAddressModeClamp;
      dt_rz.filterMode = cudaFilterModePoint;
      
      CUDA_SAFE_CALL( cudaBindTextureToArray( dt_rx, this->d_rx.GetArray() ) );
      CUDA_SAFE_CALL( cudaBindTextureToArray( dt_ry, this->d_ry.GetArray() ) );
      CUDA_SAFE_CALL( cudaBindTextureToArray( dt_rz, this->d_rz.GetArray() ) );
      

      // Run the kernel
      dim3 grid, threads;

      threads.x = threads.y = kCMPKernelSize;
      threads.z = 1;

      grid = this->d_rx.CoverBlocks( kCMPKernelSize );
      grid.z = 1;

      GCAmorphGPU::tCMPcompute.Start();
      CompMetPropKernel<<<grid,threads>>>
	( this->d_origArea, this->d_invalid,
	  this->d_area, this->d_area1, this->d_area2,
	  d_globals );
      CUDA_CHECK_ERROR( "CompMetPropKernel failed!\n" );
      GCAmorphGPU::tCMPcompute.Stop();

      // Retrieve global statistics
      int globals[2];
      CUDA_SAFE_CALL( cudaMemcpy( &globals, d_globals,
				  2*sizeof(int),
				  cudaMemcpyDeviceToHost ) );
      invalid = globals[iCMPGlobalsInvalid];
      this->neg = globals[iCMPGlobalsNeg];

      // Release device temporary
      CUDA_SAFE_CALL( cudaFree( d_globals ) );

      // Unbind the textures
      CUDA_SAFE_CALL( cudaUnbindTexture( dt_rx ) );
      CUDA_SAFE_CALL( cudaUnbindTexture( dt_ry ) );
      CUDA_SAFE_CALL( cudaUnbindTexture( dt_rz ) );



      GCAmorphGPU::tCMPtot.Stop();
    }


    // --------------------------------------------



    void GCAmorphGPU::ClearGradient( void ) {
      this->d_dx.Zero();
      this->d_dy.Zero();
      this->d_dz.Zero();
    }

    void GCAmorphGPU::ClearMomentum( void ) {
      this->d_odx.Zero();
      this->d_ody.Zero();
      this->d_odz.Zero();
    }

    
    // --------------------------------------------

    const unsigned int kApplyGradientKernelSize = 16;

    __device__ void FetchDerivs( const unsigned int ix,
				 const unsigned int iy,
				 const unsigned int iz,
				 float& dx, float& dy, float& dz ) {

      const float xLoc = ix+0.5f;
      const float yLoc = iy+0.5f;
      const float zLoc = iz+0.5f;

      dx = tex3D( dt_dx, xLoc, yLoc, zLoc );
      dy = tex3D( dt_dy, xLoc, yLoc, zLoc );
      dz = tex3D( dt_dz, xLoc, yLoc, zLoc );
    }

    __global__
    void ApplyGradientKernel( const VolumeArgGPU<char> invalid,
			      VolumeArgGPU<float> odx,
			      VolumeArgGPU<float> ody,
			      VolumeArgGPU<float> odz,
			      VolumeArgGPU<float> rx,
			      VolumeArgGPU<float> ry,
			      VolumeArgGPU<float> rz,
			      const float dt, const float momentum ) {
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;

      for( unsigned int iz = 0; iz< invalid.dims.z; iz++ ) {
	if( invalid.InVolume(ix,iy,iz) ) {

	  if( invalid(ix,iy,iz) == GCAM_POSITION_INVALID ) {
	    continue;
	  }

	  // Fetch the dx, dy and dz values from textures
	  float gcamdx, gcamdy, gcamdz;
	  FetchDerivs( ix, iy, iz, gcamdx, gcamdy, gcamdz );

	  float ldx, ldy, ldz;

	  ldx = gcamdx*dt + odx(ix,iy,iz)*momentum;
	  ldy = gcamdy*dt + ody(ix,iy,iz)*momentum;
	  ldz = gcamdz*dt + odz(ix,iy,iz)*momentum;

	  // Update odx, ody, odz
	  odx(ix,iy,iz) = ldx;
	  ody(ix,iy,iz) = ldy;
	  odz(ix,iy,iz) = ldz;

	  // Update x, y z
	  rx(ix,iy,iz) += ldx;
	  ry(ix,iy,iz) += ldy;
	  rz(ix,iy,iz) += ldz;
	}
      }
    }

    void GCAmorphGPU::ApplyGradient( GCA_MORPH_PARMS *parms ) {
      
      // Start with a sanity check
      this->CheckIntegrity();

      // Put dx, dy and dz into textures
      this->d_dx.AllocateArray();
      this->d_dy.AllocateArray();
      this->d_dz.AllocateArray();
      this->d_dx.SendArray();
      this->d_dy.SendArray();
      this->d_dz.SendArray();

      dt_dx.normalized = false;
      dt_dx.addressMode[0] = cudaAddressModeClamp;
      dt_dx.addressMode[1] = cudaAddressModeClamp;
      dt_dx.addressMode[2] = cudaAddressModeClamp;
      dt_dx.filterMode = cudaFilterModePoint;

      dt_dy.normalized = false;
      dt_dy.addressMode[0] = cudaAddressModeClamp;
      dt_dy.addressMode[1] = cudaAddressModeClamp;
      dt_dy.addressMode[2] = cudaAddressModeClamp;
      dt_dy.filterMode = cudaFilterModePoint;

      dt_dz.normalized = false;
      dt_dz.addressMode[0] = cudaAddressModeClamp;
      dt_dz.addressMode[1] = cudaAddressModeClamp;
      dt_dz.addressMode[2] = cudaAddressModeClamp;
      dt_dz.filterMode = cudaFilterModePoint;
      
      CUDA_SAFE_CALL( cudaBindTextureToArray( dt_dx, this->d_dx.GetArray() ) );
      CUDA_SAFE_CALL( cudaBindTextureToArray( dt_dy, this->d_dy.GetArray() ) );
      CUDA_SAFE_CALL( cudaBindTextureToArray( dt_dz, this->d_dz.GetArray() ) );
      
      

      // Run the computation
      dim3 grid, threads;
      
      threads.x = threads.y = kApplyGradientKernelSize;
      threads.z = 1;

      grid = this->d_invalid.CoverBlocks( kApplyGradientKernelSize );
      grid.z = 1;

      ApplyGradientKernel<<<grid,threads>>>
	( this->d_invalid,
	  this->d_odx, this->d_ody, this->d_odz,
	  this->d_rx, this->d_ry, this->d_rz,
	  parms->dt, parms->momentum );
      CUDA_CHECK_ERROR( "ApplyGradientKernel failed!\n" );


      // Unbind the textures
      CUDA_SAFE_CALL( cudaUnbindTexture( dt_dx ) );
      CUDA_SAFE_CALL( cudaUnbindTexture( dt_dy ) );
      CUDA_SAFE_CALL( cudaUnbindTexture( dt_dz ) );


      // Something we can't do yet....
      if (!DZERO(parms->l_area_intensity)) {
	std::cerr << __FUNCTION__ 
		  << ": gcamCreateNodeLookupTable not implemented!"
		  << std::endl;
	exit( EXIT_FAILURE );
      }

    }

    // --------------------------------------------
    
    const unsigned int kUndoGradientKernelSize = 16;
    
    __global__
    void UndoGradientKernel( const VolumeArgGPU<char> invalid,
			     VolumeArgGPU<float> odx,
			     VolumeArgGPU<float> ody,
			     VolumeArgGPU<float> odz,
			     VolumeArgGPU<float> rx,
			     VolumeArgGPU<float> ry,
			     VolumeArgGPU<float> rz ) {
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;
      
      for( unsigned int iz = 0; iz< invalid.dims.z; iz++ ) {
	if( invalid.InVolume(ix,iy,iz) ) {
	  
	  if( invalid(ix,iy,iz) == GCAM_POSITION_INVALID ) {
	    continue;
	  }
	  
	  float ldx = odx(ix,iy,iz);
	  float ldy = ody(ix,iy,iz);
	  float ldz = odz(ix,iy,iz);
	  
	  // Update odx, ody, odz
	  odx(ix,iy,iz) = 0;
	  ody(ix,iy,iz) = 0;
	  odz(ix,iy,iz) = 0;
	  
	  // Update x, y z
	  rx(ix,iy,iz) -= ldx;
	  ry(ix,iy,iz) -= ldy;
	    rz(ix,iy,iz) -= ldz;
	}
      }
    }


    void GCAmorphGPU::UndoGradient( void ) {

      this->CheckIntegrity();
      
      // Run the computation
      dim3 grid, threads;
      
      threads.x = threads.y = kUndoGradientKernelSize;
      threads.z = 1;

      grid = this->d_invalid.CoverBlocks( kUndoGradientKernelSize );
      grid.z = 1;

      UndoGradientKernel<<<grid,threads>>>
	( this->d_invalid,
	  this->d_odx, this->d_ody, this->d_odz,
	  this->d_rx, this->d_ry, this->d_rz );
      CUDA_CHECK_ERROR( "UndoGradientKernel failed!\n" );
    }

    // --------------------------------------------

    const unsigned int kAddStatusKernelSize = 16;

    __global__
    void AddStatusKernel( VolumeArgGPU<int> status, const int addState ) {
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;
      
      for( unsigned int iz = 0; iz< status.dims.z; iz++ ) {
	if( status.InVolume(ix,iy,iz) ) {
	  status(ix,iy,iz) |= addState;
	}
      }
    }

    void GCAmorphGPU::AddStatus( const int addState ) {

      this->CheckIntegrity();

      // Run the computation
      dim3 grid, threads;
      
      threads.x = threads.y = kAddStatusKernelSize;
      threads.z = 1;

      grid = this->d_status.CoverBlocks( kAddStatusKernelSize );
      grid.z = 1;
      
      AddStatusKernel<<<grid,threads>>>( this->d_status, addState );
      CUDA_CHECK_ERROR( "AddStatusKernel failed!" );
    }


    // --------------------------------------------

    const unsigned int kRemoveStatusKernelSize = 16;

    __global__
    void RemoveStatusKernel( VolumeArgGPU<int> status, const int subtractState ) {
      const unsigned int bx = ( blockIdx.x * blockDim.x );
      const unsigned int by = ( blockIdx.y * blockDim.y );
      const unsigned int ix = threadIdx.x + bx;
      const unsigned int iy = threadIdx.y + by;
      
      const int invState = ~subtractState;

      for( unsigned int iz = 0; iz< status.dims.z; iz++ ) {
	if( status.InVolume(ix,iy,iz) ) {
	  status(ix,iy,iz) &= invState;
	}
      }
    }

    void GCAmorphGPU::RemoveStatus( const int subtractState ) {

      this->CheckIntegrity();

      // Run the computation
      dim3 grid, threads;
      
      threads.x = threads.y = kRemoveStatusKernelSize;
      threads.z = 1;

      grid = this->d_status.CoverBlocks( kRemoveStatusKernelSize );
      grid.z = 1;
      
      RemoveStatusKernel<<<grid,threads>>>( this->d_status, subtractState );
      CUDA_CHECK_ERROR( "AddStatusKernel failed!" );
    }
    
    // --------------------------------------------

    void GCAmorphGPU::ResetLabelNodeStatus( void ) {
      this->RemoveStatus( GCAM_LABEL_NODE );
      this->RemoveStatus( GCAM_IGNORE_LIKELIHOOD );
    }
    

    // ----------------------------------------------------
    void GCAmorphGPU::ShowTimings( void ) {
#ifdef CUDA_SHOW_TIMINGS
      std::cout << "==================================" << std::endl;
      std::cout << "GCAmorphGPU timers" << std::endl;
      std::cout << "------------------" << std::endl;
#ifndef CUDA_FORCE_SYNC
      std::cout << "WARNING: CUDA_FORCE_SYNC not #defined" << std::endl;
      std::cout << "Timings may not be accurate" << std::endl;
#endif
      std::cout << std::endl;

      std::cout << "Send:" << std::endl;
      std::cout << "      Pack : " << GCAmorphGPU::tSendPack << std::endl;
      std::cout << "  Transfer : " << GCAmorphGPU::tSendTransfer << std::endl;
      std::cout << "Total      : " << GCAmorphGPU::tSendTot << std::endl;
      std::cout << std::endl;

      std::cout << "Recv:" << std::endl;
      std::cout << "      Pack : " << GCAmorphGPU::tRecvPack << std::endl;
      std::cout << "  Transfer : " << GCAmorphGPU::tRecvTransfer << std::endl;
      std::cout << "Total      : " << GCAmorphGPU::tRecvTot << std::endl;
      std::cout << std::endl;

      std::cout << "Host Memory:" << std::endl;
      std::cout << "     Alloc : " << GCAmorphGPU::tHostAlloc << std::endl;
      std::cout << "   Release : " << GCAmorphGPU::tHostRelease << std::endl;
      std::cout << " Randomise : " << GCAmorphGPU::tHostRandomise << std::endl;
      std::cout << std::endl;

      std::cout << "Compute Metric Properties:" << std::endl;
      std::cout << "   Compute : " << GCAmorphGPU::tCMPcompute << std::endl;
      std::cout << "Total      : " << GCAmorphGPU::tCMPtot << std::endl;
      std::cout << std::endl;

      std::cout << "==================================" << std::endl;
#endif
    }
    

    
    // Define static members
    SciGPU::Utilities::Chronometer GCAmorphGPU::tSendTot;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tSendPack;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tSendTransfer;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tRecvTot;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tRecvPack;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tRecvTransfer;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tHostAlloc;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tHostRelease;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tHostRandomise;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tCMPtot;
    SciGPU::Utilities::Chronometer GCAmorphGPU::tCMPcompute;


    dim3 GCAmorphGPU::hostDims = make_uint3(0,0,0);
    float *GCAmorphGPU::h_rx, *GCAmorphGPU::h_ry, *GCAmorphGPU::h_rz;
    float *GCAmorphGPU::h_origx, *GCAmorphGPU::h_origy, *GCAmorphGPU::h_origz;
    float *GCAmorphGPU::h_dx, *GCAmorphGPU::h_dy, *GCAmorphGPU::h_dz;
    float *GCAmorphGPU::h_odx, *GCAmorphGPU::h_ody, *GCAmorphGPU::h_odz;
    float *GCAmorphGPU::h_origArea, *GCAmorphGPU::h_origArea1, *GCAmorphGPU::h_origArea2;
    float *GCAmorphGPU::h_area, *GCAmorphGPU::h_area1, *GCAmorphGPU::h_area2;
    char *GCAmorphGPU::h_invalid;
    int *GCAmorphGPU::h_label, *GCAmorphGPU::h_status;
    float *GCAmorphGPU::h_labelDist;
    float *GCAmorphGPU::h_mean;
    float *GCAmorphGPU::h_variance;



    void GCAmorphGPU::AllocateHost( const GCAmorphGPU& gcam ) {
      
      // Check integrity
      gcam.CheckIntegrity();

      // Check if current allocation OK
      const dim3 gcamDims = gcam.d_rx.GetDims();
      const size_t reqSize = gcamDims.x * gcamDims.y * gcamDims.z;
      size_t currSize = GCAmorphGPU::hostDims.x * GCAmorphGPU::hostDims.y * GCAmorphGPU::hostDims.z;

      if( reqSize <= currSize ) {
	return;
      }

      std::cerr << __FUNCTION__
		<< ": Warning - not thread safe!" << std::endl;

      // Get rid of the old allocation
      GCAmorphGPU::ReleaseHost();

      GCAmorphGPU::tHostAlloc.Start();
      // Set dimensions
      GCAmorphGPU::hostDims = gcam.d_rx.GetDims();

      // Do the allocations
      GCAmorphGPU::h_rx = gcam.d_rx.AllocateHostBuffer();
      GCAmorphGPU::h_ry = gcam.d_ry.AllocateHostBuffer();
      GCAmorphGPU::h_rz = gcam.d_rz.AllocateHostBuffer();
     
      GCAmorphGPU::h_origx = gcam.d_origx.AllocateHostBuffer();
      GCAmorphGPU::h_origy = gcam.d_origy.AllocateHostBuffer();
      GCAmorphGPU::h_origz = gcam.d_origz.AllocateHostBuffer();

      GCAmorphGPU::h_dx = gcam.d_dx.AllocateHostBuffer();
      GCAmorphGPU::h_dy = gcam.d_dy.AllocateHostBuffer();
      GCAmorphGPU::h_dz = gcam.d_dz.AllocateHostBuffer();

      GCAmorphGPU::h_odx = gcam.d_odx.AllocateHostBuffer();
      GCAmorphGPU::h_ody = gcam.d_ody.AllocateHostBuffer();
      GCAmorphGPU::h_odz = gcam.d_odz.AllocateHostBuffer();

      GCAmorphGPU::h_origArea = gcam.d_origArea.AllocateHostBuffer();
      GCAmorphGPU::h_origArea1 = gcam.d_origArea1.AllocateHostBuffer();
      GCAmorphGPU::h_origArea2 = gcam.d_origArea2.AllocateHostBuffer();

      GCAmorphGPU::h_area = gcam.d_area.AllocateHostBuffer();
      GCAmorphGPU::h_area1 = gcam.d_area1.AllocateHostBuffer();
      GCAmorphGPU::h_area2 = gcam.d_area2.AllocateHostBuffer();

      GCAmorphGPU::h_invalid = gcam.d_invalid.AllocateHostBuffer();
      GCAmorphGPU::h_status = gcam.d_status.AllocateHostBuffer();
      GCAmorphGPU::h_label = gcam.d_label.AllocateHostBuffer();
      GCAmorphGPU::h_labelDist = gcam.d_labelDist.AllocateHostBuffer();

      GCAmorphGPU::h_mean = gcam.d_mean.AllocateHostBuffer();
      GCAmorphGPU::h_variance = gcam.d_variance.AllocateHostBuffer();

      GCAmorphGPU::tHostAlloc.Stop();

    }


    
    void GCAmorphGPU::ReleaseHost( void ) {

      // Sanity check
      if( GCAmorphGPU::hostDims == make_uint3(0,0,0) ) {
	return;
      }

      std::cerr << __FUNCTION__
		<< ": Warning - not thread safe!" << std::endl;

      GCAmorphGPU::tHostRelease.Start();

      GCAmorphGPU::hostDims = make_uint3(0,0,0);

      // Release page-locked host memory
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_rx ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_ry ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_rz ) );

      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origx ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origy ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origz ) );

      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_dx ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_dy ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_dz ) );

      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_odx ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_ody ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_odz ) );

      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origArea ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origArea1 ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origArea2 ) );

      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_area ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_area1 ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_area2 ) );

      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_invalid ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_status ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_label ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_labelDist ) );

      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_mean ) );
      CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_variance ) );

      GCAmorphGPU::tHostRelease.Stop();
    }


    

    template<typename T>
    void RandomArray( T* arr, const size_t nVals ) {
      
      for( unsigned int i=0; i<nVals; i++ ) {
	char randVal = std::rand() % std::numeric_limits<char>::max();
	arr[i] = static_cast<T>(randVal);
      }

    }


    void GCAmorphGPU::RandomiseHost( void ) {

      // Sanity check
      if( GCAmorphGPU::hostDims == make_uint3(0,0,0) ) {
	return;
      }

      std::cerr << __FUNCTION__
		<< ": Warning - not thread safe!" << std::endl;

      GCAmorphGPU::tHostRandomise.Start();

      size_t currSize = GCAmorphGPU::hostDims.x *
	GCAmorphGPU::hostDims.y * GCAmorphGPU::hostDims.z;

      RandomArray( GCAmorphGPU::h_rx, currSize );
      RandomArray( GCAmorphGPU::h_ry, currSize );
      RandomArray( GCAmorphGPU::h_rz, currSize );

      
      RandomArray( GCAmorphGPU::h_origx, currSize );
      RandomArray( GCAmorphGPU::h_origy, currSize );
      RandomArray( GCAmorphGPU::h_origz, currSize );

      RandomArray( GCAmorphGPU::h_dx, currSize );
      RandomArray( GCAmorphGPU::h_dy, currSize );
      RandomArray( GCAmorphGPU::h_dz, currSize );

      RandomArray( GCAmorphGPU::h_odx, currSize );
      RandomArray( GCAmorphGPU::h_ody, currSize );
      RandomArray( GCAmorphGPU::h_odz, currSize );

      RandomArray( GCAmorphGPU::h_origArea, currSize );
      RandomArray( GCAmorphGPU::h_origArea1, currSize );
      RandomArray( GCAmorphGPU::h_origArea2, currSize );

      RandomArray( GCAmorphGPU::h_area, currSize );
      RandomArray( GCAmorphGPU::h_area1, currSize );
      RandomArray( GCAmorphGPU::h_area2, currSize );

      RandomArray( GCAmorphGPU::h_invalid, currSize );
      RandomArray( GCAmorphGPU::h_status, currSize );
      RandomArray( GCAmorphGPU::h_label, currSize );
      RandomArray( GCAmorphGPU::h_labelDist, currSize );

      RandomArray( GCAmorphGPU::h_mean, currSize );
      RandomArray( GCAmorphGPU::h_variance, currSize );

      
      GCAmorphGPU::tHostRandomise.Stop();
    }

  }
}


void gcamClearGradientGPU( GCA_MORPH* gcam ) {
  GPU::Classes::GCAmorphGPU gcamGPU;
  
  gcamGPU.SendAll( gcam );
  gcamGPU.ClearGradient();
  gcamGPU.RecvAll( gcam );

}

void gcamClearMomentumGPU( GCA_MORPH* gcam ) {
  GPU::Classes::GCAmorphGPU gcamGPU;
  
  gcamGPU.SendAll( gcam );
  gcamGPU.ClearMomentum();
  gcamGPU.RecvAll( gcam );

}


void gcamComputeMetricPropertiesGPU( GCA_MORPH* gcam,
				     int *invalid ) {
  /*!
    This is a wrapper around the CUDA implementation
    of gcamComputeMetricProperties
  */

  GPU::Classes::GCAmorphGPU gcamGPU;
  
  gcamGPU.SendAll( gcam );
  gcamGPU.ComputeMetricProperties( *invalid );
  gcamGPU.RecvAll( gcam );

}


void gcamApplyGradientGPU( GCA_MORPH *gcam, GCA_MORPH_PARMS *parms ) {

  GPU::Classes::GCAmorphGPU gcamGPU;
  
  gcamGPU.SendAll( gcam );
  gcamGPU.ApplyGradient( parms );
  gcamGPU.RecvAll( gcam );
}


void gcamUndoGradientGPU( GCA_MORPH *gcam ) {

  GPU::Classes::GCAmorphGPU gcamGPU;
  
  gcamGPU.SendAll( gcam );
  gcamGPU.UndoGradient();
  gcamGPU.RecvAll( gcam );
}


void gcamAddStatusGPU( GCA_MORPH *gcam, const int statusFlags ) {

  GPU::Classes::GCAmorphGPU gcamGPU;
  
  gcamGPU.SendAll( gcam );
  gcamGPU.AddStatus( statusFlags );
  gcamGPU.RecvAll( gcam );
}


void gcamRemoveStatusGPU( GCA_MORPH *gcam, const int statusFlags ) {

  GPU::Classes::GCAmorphGPU gcamGPU;
  
  gcamGPU.SendAll( gcam );
  gcamGPU.RemoveStatus( statusFlags );
  gcamGPU.RecvAll( gcam );
}

#endif
