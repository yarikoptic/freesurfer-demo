#include "hip/hip_runtime.h"
/**
 * @file  em_register_cuda.cu
 * @brief Holds em_register CUDA routines
 *
 * Contains CUDA routines for em_register
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2010/08/26 00:00:55 $
 *    $Revision: 1.3.2.1 $
 *
 * Copyright (C) 2002-2008,
 * The General Hospital Corporation (Boston, MA). 
 * All rights reserved.
 *
 * Distribution, usage and copying of this software is covered under the
 * terms found in the License Agreement file named 'COPYING' found in the
 * FreeSurfer source code root directory, and duplicated here:
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferOpenSourceLicense
 *
 * General inquiries: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include <iostream>
#include <iomanip>
using namespace std;


#include <thrust/device_new_allocator.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>

#include "cudacheck.h"

#include "mriframegpu.hpp"
#include "affinegpu.hpp"
#include "gcasgpu.hpp"

#include "generators.hpp"
#include "cudatypeutils.hpp"

#include "em_register_cuda.h"

// ==================================================================




static GPU::Classes::MRIframeGPU<unsigned char> src_uchar;
texture<unsigned char, 3, hipReadModeElementType> dt_mri;  // 3D texture

static GPU::Classes::GCASampleGPU myGCAS;

const unsigned int  kCalcLogPKernelSize = 256;
const unsigned int kOptimiseBlockSize = 256;


const unsigned int nIndices = 9;
enum Indices{ iMinTrans=0, iMaxTrans=1, inTrans=2,
	      iMinScale=3, iMaxScale=4, inScale=5,
	      iMinRot=6, iMaxRot=7, inRot=8 };

__constant__ float dc_TransformParams[nIndices];





// =================================================================
// Device Utility functions


//! Extracts data from the MRI texture
__device__ float MRIlookup( const float3 r ) {
  /*!
    Performs a lookup into the MRI texture
  */

  // Offset by 0.5, since texture values are assumed to be at voxel centres

  return( tex3D( dt_mri, r.x+0.5f, r.y+0.5f, r.z+0.5f ) );
}



//! Computes the log_p value for a single point.
__device__ float ComputeLogP( const float val, const float mean,
			      const float prior, const float covar ) {
  
  float det = covar;

  float v = val - mean;

  float log_p = - logf( sqrtf( det ) ) - 0.5*( v*v / covar ) + logf( prior );

  if( log_p < -3 ) {
    log_p = -3;
  }

  return( log_p );
}



//! Routine to sum all the logps for a given transform in shared memory
__device__ float SumLogPs( const GPU::Classes::AffineTransShared &afTrans,
			   const GPU::Classes::GCASonGPU& gcas ) {
  // The accumulator array for this block
  __shared__ float myLogps[kOptimiseBlockSize];
  myLogps[threadIdx.x] = 0;
  
  // Accumulate log p values in shared memory
  for( unsigned int i=0; i<gcas.nSamples; i+= kOptimiseBlockSize ) {
    if( (i + threadIdx.x) < gcas.nSamples ) {
      float3 rOut = afTrans.transform( gcas.GetLocation( i+threadIdx.x ) );

      float mriVal = MRIlookup( rOut );

      myLogps[threadIdx.x] += ComputeLogP( mriVal,
					   gcas.means[i+threadIdx.x],
					   gcas.priors[i+threadIdx.x],
					   gcas.covars[i+threadIdx.x] );
    }
  }

  __syncthreads();

  // Perform reduction sum
#if 0
  // Slow but always correct version
  for( unsigned int d=blockDim.x / 2; d>0; d>>=1 ) {
    if( threadIdx.x < d ) { 
      myLogps[threadIdx.x] += myLogps[threadIdx.x+d];
    }
    __syncthreads();
  }
#else
  // Version optimised for a warpsize of 32
  for( unsigned int d=blockDim.x / 2; d>32; d>>=1 ) {
    if( threadIdx.x < d ) { 
      myLogps[threadIdx.x] += myLogps[threadIdx.x+d];
    }
    __syncthreads();
  }

  if( threadIdx.x < 32 ) {
    myLogps[threadIdx.x] += myLogps[threadIdx.x+32];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+16];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+8];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+4];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+2];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+1];
  }
#endif

  return( myLogps[0] );
}




// ===================================================================
// Device Kernels


//! Kernel to compute log_p for a single transform array
__global__
void ComputeAllLogP( const GPU::Classes::AffineTransformation afTrans,
		     const GPU::Classes::GCASonGPU gcas,
		     float *logps ) {
  /*!
    Driver kernel to compute the value of log_p for every sample.
    Uses the MRI texture and transform matrix stored in constant memory
  */

  const unsigned int iSample = (blockIdx.x*blockDim.x) + threadIdx.x;

  if( iSample >= gcas.nSamples ) {
    // Nothing to do
    return;
  }

  // Compute location with affine transform
  float3 rOut = afTrans.transform( gcas.GetLocation( iSample ) );

  float mriVal = MRIlookup( rOut );


  logps[iSample] = ComputeLogP( mriVal,
				gcas.means[iSample],
				gcas.priors[iSample],
				gcas.covars[iSample] );
}


//! Kernel to compute all probabilities for a given translation generator
__global__
void TranslationLogps( const GPU::Classes::AffineTransformation base,
				  const TranslationGenerator tGen,
				  const GPU::Classes::GCASonGPU gcas,
				  float *logps ) {

  // Find our translation
  float3 myTrans = tGen( blockIdx.x );
  
  __shared__ float m1[GPU::Classes::AffineTransShared::kMatrixSize];
  __shared__ float m2[GPU::Classes::AffineTransShared::kMatrixSize];

  GPU::Classes::AffineTransShared final( m1 ), translation( m2 );

  // Recast the input transformation (which will be in shared memory)
  // as an AffineTransShared. Unfortunately, this needs a const_cast
  const float* tmp = base.GetPointer();
  const GPU::Classes::AffineTransShared bAff( const_cast<float*>(tmp) );

  // Make identity transform
  translation.SetIdentity();

  // Note that we invert the translation
  translation.SetTranslation( -myTrans );

  // Reverse order of multiplications, as compared to mri_em_register.c
  final.Multiply( bAff, translation );
  
  __syncthreads();

  // -- All threads now have access to the transformation

  
  // Compute the final result

  float myLogP = SumLogPs( final, gcas );

  if( threadIdx.x == 0 ) {
    logps[ blockIdx.x ] = myLogP;
  }
}



//! Kernel to compute all transform probabilities for a given transform generator
__global__
void TransformLogps( const GPU::Classes::AffineTransformation base,
		     const float3 originTranslation,
		     const GPU::Classes::GCASonGPU gcas,
		     float *logps ) {

  
  const long b1d = blockIdx.x + ( blockIdx.y * gridDim.x );

  // Find our transform
  LinearGenerator translate( dc_TransformParams[iMinTrans],
			     dc_TransformParams[iMaxTrans],
			     dc_TransformParams[inTrans] );
  LinearGenerator scale( dc_TransformParams[iMinScale],
			 dc_TransformParams[iMaxScale],
			 dc_TransformParams[inScale] );
  LinearGenerator rotate( dc_TransformParams[iMinRot],
			  dc_TransformParams[iMaxRot],
			  dc_TransformParams[inRot] );

  TransformGenerator tGen( translate, scale, rotate );

  float3 myTrans, myScale, myRot;

  tGen.GetTransform( b1d, myTrans, myScale, myRot );


  // Invert the transforms
  myTrans = -myTrans;
  myScale.x = 1/myScale.x;
  myScale.y = 1/myScale.y;
  myScale.z = 1/myScale.z;
  myRot = -myRot;


  // Compute the transform matrix
  __shared__ float m1[GPU::Classes::AffineTransShared::kMatrixSize];
  __shared__ float m2[GPU::Classes::AffineTransShared::kMatrixSize];
  __shared__ float m3[GPU::Classes::AffineTransShared::kMatrixSize];

  GPU::Classes::AffineTransShared A( m1 ), B( m2 ), C( m3 ) ;

  // Recast the input transformations (which will be in shared memory)
  // as an AffineTransShared. Unfortunately, this needs a const_cast
  const float* tmp = base.GetPointer();
  const GPU::Classes::AffineTransShared bAff( const_cast<float*>(tmp) );
  
  // Invert the order in the original find_optimal_linear_xform routine
  B.SetIdentity();
  B.SetTranslation( originTranslation );

  A.Multiply( bAff, B );
  
  B.SetIdentity();
  B.SetXRotation( myRot.x );
  
  C.Multiply( A, B );

  A.SetIdentity();
  A.SetYRotation( myRot.y );

  B.Multiply( C, A );

  A.SetIdentity();
  A.SetZRotation( myRot.z );
  
  C.Multiply( B, A );

  A.SetIdentity();
  A.SetScaling( myScale );

  B.Multiply( C, A );

  A.SetIdentity();
  A.SetTranslation( -originTranslation );

  C.Multiply( B, A );

  A.SetIdentity();
  A.SetTranslation( myTrans );

  B.Multiply( C, A );

  __syncthreads();

  // -- All threads now have access to the transformation
  
  float myLogp = SumLogPs( B, gcas );

  // Write the final result
  if( threadIdx.x == 0 ) {
    logps[ b1d ] = myLogp;
  }
}





// ===================================================================
// External Functions

float CUDA_ComputeLogSampleProbability( const MATRIX *m_L ) {
/*!
    Re-implementation of local_GCAcomputeLogSampleProbability() from
    file mri_em_register.c.
    Assumes that robust is set to false, making the original routine
    a pass-through to GCAcomputeLogSampleProbability() from file
    gca.c.
    Also assumed that CUDA_em_register_Prepare has already
    set up everything
  */

  thrust::device_ptr<float> d_logpvals;
  MATRIX *inv_m_L = NULL;

  // Get the inverse of the transform matrix
  inv_m_L = MatrixInverse( (MATRIX*)m_L, inv_m_L );

  GPU::Classes::AffineTransformation myTransform( inv_m_L );
  const GPU::Classes::GCASonGPU myGCASonGPU( myGCAS );
  const unsigned int nsamples = myGCASonGPU.nSamples;

  d_logpvals = thrust::device_new<float>( nsamples );

  // ---------------------------------------
  float logps;

  // Run the log_p evaluation kernel
  dim3 grid, threads;
  threads.x = kCalcLogPKernelSize;
  threads.y = threads.z = 1;
  grid.x = static_cast<int>( ceilf ( static_cast<float>(nsamples) / threads.x ) );
  grid.y = grid.z = 1;

  ComputeAllLogP<<<grid,threads>>>( myTransform, myGCASonGPU,
				    thrust::raw_pointer_cast(d_logpvals) );
  CUDA_CHECK_ERROR( "ComputeAllLogP kernel failed!\n" );

  // Do the reduction
  logps = thrust::reduce( d_logpvals, d_logpvals+nsamples );

  MatrixFree( &inv_m_L );
  thrust::device_delete( d_logpvals );

  return( logps );
}





void CUDA_FindOptimalTranslation( const MATRIX *baseTransform,
				  const float minTrans,
				  const float maxTrans,
				  const unsigned int nTrans,
				  float *maxLogP,
				  float *dx,
				  float *dy,
				  float *dz ) {
  /*!
    Routine to find the best translation to match the
    MRI to the classifier array.
    A 'base' transform is supplied, and then translations
    within the given limits are searched
  */

  const unsigned int totalTrans = nTrans * nTrans * nTrans;

  // Device vector to hold logps
  thrust::device_ptr<float> d_logps;

  d_logps = thrust::device_new<float>( totalTrans );

  // Construct the generator which will give the required translations
  TranslationGenerator myGen( minTrans, maxTrans, nTrans );
  
  

  // Extract the 'base' transform, inverting
  MATRIX *invBaseTransform = NULL;
  invBaseTransform = MatrixInverse( baseTransform, invBaseTransform );
  GPU::Classes::AffineTransformation myBaseTransform( invBaseTransform );

  // Get the GCAsample array, which must already be on the GPU
  const GPU::Classes::GCASonGPU myGCASonGPU( myGCAS );

  // Compute all the probabilities
  dim3 grid, threads;

  threads.x = kOptimiseBlockSize;
  threads.y = threads.z = 1;

  grid.x = totalTrans;
  grid.y = grid.z = 1;

  TranslationLogps<<<grid,threads>>>( myBaseTransform,
				      myGen,
				      myGCAS,
				      thrust::raw_pointer_cast( d_logps ) );
  CUDA_CHECK_ERROR( "TranslationLogps failed!" );
 
#if 0
  for( unsigned int i=0; i<totalTrans; i++ ) {
    float3 translation = myGen(i);
    cout << __FUNCTION__  << " "
	 << setw(8) << setprecision(4) << translation.x << " "
	 << setw(8) << setprecision(4) << translation.y << " "
	 << setw(8) << setprecision(4) << translation.z << " "
	 << setw(12) << setprecision(8) << d_logps[i] << endl;
  }
#endif

  // Extract the maximum location
  thrust::device_ptr<float> maxLoc;
  maxLoc = thrust::max_element( d_logps, d_logps+totalTrans );

  // Get the maximum value
  *maxLogP = *maxLoc;

  // Convert the location to the required translation
  const int index = (maxLoc - d_logps);
  const float3 trans = myGen(index);


  *dx = trans.x;
  *dy = trans.y;
  *dz = trans.z;

  thrust::device_delete( d_logps );
}





void CUDA_FindOptimalTransform( const MATRIX *baseTransform,
				const MATRIX *originTranslation,
				const float minTrans,
				const float maxTrans,
				const unsigned int nTrans,
				const float minScale,
				const float maxScale,
				const unsigned nScale,
				const float minRot,
				const float maxRot,
				const unsigned int nRot,
				double *maxLogP,
				double *dx,
				double *dy,
				double *dz,
				double *sx,
				double *sy,
				double *sz,
				double *rx,
				double *ry,
				double *rz ) {
  /*!
    Routine to find the best transformation to match the
    MRI to the classifier array.
    A 'base' transform is supplied, and then transforms
    within the given limits are searched
  */

  const unsigned int totTranslate = nTrans * nTrans * nTrans;
  const unsigned int totOther = nScale * nScale * nScale *
    nRot * nRot * nRot;

  const unsigned int totalTransforms = totTranslate * totOther;

  // Device vector to hold logps
  thrust::device_ptr<float> d_logps;

  d_logps = thrust::device_new<float>( totalTransforms );

  // Construct the generator which will give the required translations
  LinearGenerator translate( minTrans, maxTrans, nTrans );
  LinearGenerator scale( minScale, maxScale, nScale );
  LinearGenerator rotate( minRot, maxRot, nRot );
  TransformGenerator myGen( translate, scale, rotate );

  // Get the transform generation parameters to the device
  float tParams[nIndices];

  tParams[iMinTrans] = minTrans;
  tParams[iMaxTrans] = maxTrans;
  tParams[inTrans] = nTrans;
  tParams[iMinScale] = minScale;
  tParams[iMaxScale] = maxScale;
  tParams[inScale] = nScale;
  tParams[iMinRot] = minRot;
  tParams[iMaxRot] = maxRot;
  tParams[inRot] = nRot;

  CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "dc_TransformParams"),
				      tParams,
				      nIndices*sizeof(float),
				      0,
				      hipMemcpyHostToDevice ) );

  // Extract the 'base' transform, inverting
  MATRIX *invBaseTransform = NULL;
  invBaseTransform = MatrixInverse( baseTransform, invBaseTransform );
  GPU::Classes::AffineTransformation myBaseTransform( invBaseTransform );

  // Extract the origin translation
  const float3 oTranslate = make_float3( originTranslation->rptr[1][4],
					 originTranslation->rptr[2][4],
					 originTranslation->rptr[3][4] );
  
  // Get the GCAsample array, which must already be on the GPU
  const GPU::Classes::GCASonGPU myGCASonGPU( myGCAS );

  dim3 grid, threads;

  // Compute all the probabilities
  threads.x = kOptimiseBlockSize;
  threads.y = threads.z = 1;

  grid.x = totTranslate;
  grid.y = totOther;
  grid.z = 1;

  TransformLogps<<<grid,threads>>>( myBaseTransform,
				    oTranslate,
				    myGCAS,
				    thrust::raw_pointer_cast( d_logps ) );
  CUDA_CHECK_ERROR( "TransformLogps failed!" );
  
  // Locate the maximum probability
  thrust::device_ptr<float> maxLoc;
  maxLoc = thrust::max_element( d_logps, d_logps+totalTransforms );

  // Extract the maximum probability
  *maxLogP = *maxLoc;


  // Convert the maximum probability to the transform parameters
  const int index = (maxLoc - d_logps);

  float3 myTrans, myScale, myRot;
  myGen.GetTransform( index, myTrans, myScale, myRot );

  *dx = myTrans.x;
  *dy = myTrans.y;
  *dz = myTrans.z;

  *sx = myScale.x;
  *sy = myScale.y;
  *sz = myScale.z;

  *rx = myRot.x;
  *ry = myRot.y;
  *rz = myRot.z;

  thrust::device_delete( d_logps );

}




// =================================================================


void CUDA_em_register_Prepare( GCA *gca,
			       GCA_SAMPLE *gcas,
			       const MRI *mri,
			       const int nSamples ) {


  // Sanity check
  if( gca->ninputs != 1 ) {
    cerr << __FUNCTION__ << ": Must have ninputs==1" << endl;
    exit( EXIT_FAILURE );
  }

  // ------------------------------
  // Send the MRI
  const unsigned int nFrame = 0;
  
  src_uchar.Allocate( mri );
  src_uchar.AllocateArray();
  src_uchar.Send( mri, nFrame );
  src_uchar.SendArray();

  // Bind to texture
  dt_mri.normalized = false;
  dt_mri.addressMode[0] = hipAddressModeClamp;
  dt_mri.addressMode[1] = hipAddressModeClamp;
  dt_mri.addressMode[2] = hipAddressModeClamp;
  dt_mri.filterMode = hipFilterModePoint;

  CUDA_SAFE_CALL( hipBindTextureToArray( dt_mri, src_uchar.GetArray() ) );

  // Send the GCAS

  myGCAS.SendGPU( gca, gcas, mri, nSamples );

}


void CUDA_em_register_Release( void ) {
  CUDA_SAFE_CALL( hipUnbindTexture( dt_mri ) );
}
